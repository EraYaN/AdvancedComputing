#include "hip/hip_runtime.h"
#include "cuda-kernels.h"

using namespace std;

__global__ void rgb2grayCudaKernel(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (x < width && y < height) {
		float r = static_cast<float>(inputImage[(y * width) + x]);
		float g = static_cast<float>(inputImage[(width * height) + (y * width) + x]);
		float b = static_cast<float>(inputImage[(2 * width * height) + (y * width) + x]);

		float grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

		grayImage[((y * width) + x)] = static_cast<unsigned char>(grayPix);
	}
}

void rgb2grayCuda(unsigned char *inputImage, unsigned char *grayImage, unsigned char *dev_grayImage, const int width, const int height, ResultContainer *result, double cpu_frequency) {
	auto t_preprocessing = now();
	auto t_init = t_preprocessing;
	// Kernel

	// specify thread and block dimensions
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(ceil((double)width / threadsPerBlock.x), ceil((double)height / threadsPerBlock.y));

	// allocate GPU memory
	unsigned char *dev_inputImage;

	//checkCudaCall(hipHostGetDevicePointer(&dev_inputImage, inputImage, 0));

	checkCudaCall(hipMalloc(&dev_inputImage, 3*width*height * sizeof(unsigned char)));

	checkCudaCall(hipMemcpy(dev_inputImage, inputImage, 3 * width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

	auto t_kernel = now();
	// execute actual function
	rgb2grayCudaKernel<<<numBlocks, threadsPerBlock>>>(dev_inputImage, dev_grayImage, width, height);
	//checkCudaCall(hipDeviceSynchronize());
	auto t_cleanup = now();

	checkCudaCall(hipMemcpy(grayImage, dev_grayImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

	checkCudaCall(hipFree(dev_inputImage));

	// /Kernel
	auto t_postprocessing = now();
	auto t_end = t_postprocessing;

	*result = ResultContainer(t_preprocessing, t_init, t_kernel, t_cleanup, t_postprocessing, t_end, cpu_frequency);
}

