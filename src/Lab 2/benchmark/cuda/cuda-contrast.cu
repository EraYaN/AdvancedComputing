#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "../checkCudaCall.h"

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;


/////////////////////////////////////
__global__ void contrast1DCudaKernel(unsigned char *grayImage, const int width, const int height, const int min, const int max, const float diff) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (x < width && y < height) {
		unsigned char pixel = grayImage[(y * width) + x];

		if (pixel < min) {
			pixel = 0;
		} else if (pixel > max) {
			pixel = 255;
		} else {
			pixel = static_cast<unsigned char>(255.0f * (pixel - min) / diff);
		}

		grayImage[(y * width) + x] = pixel;
	}
}


void contrast1DCuda(unsigned char *grayImage, const int width, const int height,
	unsigned int *histogram, const unsigned int HISTOGRAM_SIZE,
	const unsigned int CONTRAST_THRESHOLD) {
	unsigned int i = 0;
	NSTimer kernelTime = NSTimer("kernelTime", false, false);

	while ((i < HISTOGRAM_SIZE) && (histogram[i] < CONTRAST_THRESHOLD)) {
		i++;
	}
	unsigned int min = i;

	i = HISTOGRAM_SIZE - 1;
	while ((i > min) && (histogram[i] < CONTRAST_THRESHOLD)) {
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	kernelTime.start();
	// Kernel

	// specify thread and block dimensions
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

	// allocate GPU memory
	unsigned char *dev_a;
	int size = width * height;

	hipMalloc((void**)&dev_a, size * (sizeof(unsigned char)));

	// copy grayImage to GPU memory
	hipMemcpy(dev_a, grayImage, size * (sizeof(unsigned char)), hipMemcpyHostToDevice);

	// execute actual function
	contrast1DCudaKernel << <numBlocks, threadsPerBlock >> > (dev_a, width, height, min, max, diff);

	// copy result from GPU memory to grayImage
	hipMemcpy(grayImage, dev_a, size * (sizeof(unsigned char)), hipMemcpyDeviceToHost);

	// free memory
	hipFree(dev_a);

	// /Kernel
	kernelTime.stop();

	cout << fixed << setprecision(6);
	cout << "contrast1D (cpu): \t\t" << kernelTime.getElapsed() << " seconds." << endl;
}
