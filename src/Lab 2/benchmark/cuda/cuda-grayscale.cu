#include "hip/hip_runtime.h"
#include "cuda-kernels.h"

using namespace std;

__global__ void rgb2grayCudaKernel(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (x < width && y < height) {
		float r = static_cast<float>(inputImage[(y * width) + x]);
		float g = static_cast<float>(inputImage[(width * height) + (y * width) + x]);
		float b = static_cast<float>(inputImage[(2 * width * height) + (y * width) + x]);

		float grayPix = (0.3f * r) + (0.59f * g) + (0.11f * b);

		grayImage[((y * width) + x)] = static_cast<unsigned char>(grayPix);
	}
}

void rgb2grayCuda(unsigned char *inputImage, unsigned char *grayImage, const int width, const int height, ResultContainer *result, double cpu_frequency) {
	auto t_preprocessing = now();
	auto t_init = t_preprocessing;
	// Kernel

	// specify thread and block dimensions
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

	// allocate GPU memory
	unsigned char *dev_a, *dev_b;

	checkCudaCall(hipHostGetDevicePointer(&dev_a, inputImage, 0));
	checkCudaCall(hipHostGetDevicePointer(&dev_b, grayImage, 0));

	auto t_kernel = now();
	// execute actual function
	rgb2grayCudaKernel<<<numBlocks, threadsPerBlock>>>(dev_a, dev_b, width, height);
	checkCudaCall(hipDeviceSynchronize());
	auto t_cleanup = now();

	// /Kernel
	auto t_postprocessing = now();
	auto t_end = t_postprocessing;

	*result = ResultContainer(t_preprocessing, t_init, t_kernel, t_cleanup, t_postprocessing, t_end, cpu_frequency);
}

