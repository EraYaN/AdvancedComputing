#include "hip/hip_runtime.h"
#include "cuda-kernels.h"

using namespace std;


/////////////////////////////////////
__global__ void contrast1DCudaKernel(unsigned char *grayImage, const int width, const int height, const int min, const int max, const float diff) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (x < width && y < height) {
		unsigned char pixel = grayImage[(y * width) + x];

		if (pixel < min) {
			pixel = 0;
		} else if (pixel > max) {
			pixel = 255;
		} else {
			pixel = static_cast<unsigned char>(255.0f * (pixel - min) / diff);
		}

		grayImage[(y * width) + x] = pixel;
	}
}

void contrast1DCuda(unsigned char *grayImage, const int width, const int height, unsigned int *histogram, const unsigned int histogramSize, const unsigned int contrastThreshold, ResultContainer *result, double cpu_frequency) {
	auto t_preprocessing = now();
	unsigned int i = 0;

	while ((i < histogramSize) && (histogram[i] < contrastThreshold)) {
		i++;
	}
	unsigned int min = i;

	i = histogramSize - 1;
	while ((i > min) && (histogram[i] < contrastThreshold)) {
		i--;
	}
	unsigned int max = i;
	float diff = max - min;

	auto t_init = now();
	// Kernel

	// specify thread and block dimensions
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

	// allocate GPU memory
	unsigned char *dev_a;

	checkCudaCall(hipHostGetDevicePointer(&dev_a, grayImage, 0));

	//hipMalloc(&dev_a, width*height * sizeof(unsigned char));
	//hipMemcpy(dev_a, grayImage, width*height * sizeof(unsigned char), hipMemcpyHostToDevice);

	auto t_kernel = now();
	// execute actual function
	contrast1DCudaKernel<<<numBlocks, threadsPerBlock>>>(dev_a, width, height, min, max, diff);
	checkCudaCall(hipDeviceSynchronize());
	auto t_cleanup = now();

	//hipMemcpy(grayImage, dev_a, width*height * sizeof(unsigned char), hipMemcpyDeviceToHost);
	//hipFree(dev_a);

	// /Kernel
	auto t_postprocessing = now();
	auto t_end = t_postprocessing;

	*result = ResultContainer(t_preprocessing, t_init, t_kernel, t_cleanup, t_postprocessing, t_end, cpu_frequency);
}