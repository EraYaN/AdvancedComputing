#include "hip/hip_runtime.h"
#include "cuda-kernels.h"

using namespace std;

/////////////////////////////////////
__global__ void triangularSmoothCudaKernel(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height,	const float *filter)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height

	if (x < width && y < height) {
		unsigned int filterItem = 0;
		float filterSum = 0.0f;
		float smoothPix = 0.0f;

		for (int fy = y - 2; fy < y + 3; fy++) {
			for (int fx = x - 2; fx < x + 3; fx++) {
				if (((fy < 0) || (fy >= height)) || ((fx < 0) || (fx >= width))) {
					filterItem++;
					continue;
				}

				smoothPix += grayImage[(fy * width) + fx] * filter[filterItem];
				filterSum += filter[filterItem];
				filterItem++;
			}
		}

		smoothPix /= filterSum;
		smoothImage[(y * width) + x] = static_cast<unsigned char>(smoothPix);
	}
}

void triangularSmoothCuda(unsigned char *grayImage, unsigned char *smoothImage, const int width, const int height, const float *filter, double cpu_frequency) {
	auto t1 = now();
	// Kernel

	// specify thread and block dimensions
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

	// allocate GPU memory
	unsigned char *dev_a, *dev_b;
	int size = width * height;

	hipMalloc((void**)&dev_a, size * (sizeof(unsigned char)));
	hipMalloc((void**)&dev_b, size * (sizeof(unsigned char)));

	// copy grayImage to GPU memory
	hipMemcpy(dev_a, grayImage, size * (sizeof(unsigned char)), hipMemcpyHostToDevice);

	// execute actual function
	triangularSmoothCudaKernel << <numBlocks, threadsPerBlock >> > (dev_a, dev_b, width, height, filter);

	// copy result from GPU memory to grayImage
	hipMemcpy(smoothImage, dev_b, size * (sizeof(unsigned char)), hipMemcpyDeviceToHost);

	// free memory
	hipFree(dev_a);
	hipFree(dev_b);

	// /Kernel
	auto t2 = now();

	cout << fixed << setprecision(6);
	double time_elapsed = diffToNanoseconds(t1, t2, cpu_frequency);
	cout << "triangularSmooth (cpu): \t" << time_elapsed << " nanoseconds." << endl;
}